#include "hip/hip_runtime.h"
	#include "utilidades.h"
//#include "graphene_cusp.h"
#include "lattice_cusp.h"
#include <cusp/transpose.h>
#include <iostream>
#include <fstream>
#include "kpm_cusp.h"

//#define FCOMPLEX
#define DCOMPLEX
//#define FLOAT
//#define DOUBLE

#ifdef FCOMPLEX
cusp::complex<float> I(0.0f,1.0f);
cusp::complex<float> zero(0.0f,0.0f);
typedef float    FloatType;
typedef cusp::complex<float>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif

#ifdef DCOMPLEX
cusp::complex<double> I(0.0f,1.0f);
cusp::complex<double> zero(0.0f,0.0f);
typedef double    FloatType;
typedef cusp::complex<double>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif


#ifdef FLOAT
typedef float    Scalar	HCOO Vx(DDD,DDD,S*4*DDD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins	
;
typedef float    FloatType;
typedef int      Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cus0p::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<float> I(0.0f,1.0f);
float zero=0;
#endif

#ifdef DOUBLE
typedef double    Scalar;
typedef double    FloatType;
typedef int       Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<double> I(0.0,1.0);
double zero=0;
#endif


int main(int argc, char *argv[])    			//Para simplificar las corridas elegimos N=argv[1], M=argv[2]
{
    
    /***********Declaracion de apuntadores y variables del programa*********/
    int Nx, Ny,W,L,D,DD,M, ngpu,nflux,B;	//variables de conteo
	FloatType   Emin, Emax,E0min, E0max,E0;
	FloatType tU,U,p;
    //srand(1121322113);
    srand(time(0));
    
    /************Definimos los parametros iniciales **************************/
	if(argc>=12+1){
		Nx=atoi(argv[1]);                                        //Tamaño del sistema
		Ny=atoi(argv[2]);                                        //Tamaño del sistema
		M=atoi(argv[3]);
        U= atof(argv[4]);
        tU=atof(argv[5]);
		nflux  =atoi(argv[6]);
		p=atof(argv[7]);                                     //Densidad de impurezas
        L=atoi(argv[8]);
        W=atoi(argv[9]);
		E0=atof(argv[10]);                                     //Densidad de impurezas
        //      MachineName=argv[11];
        ngpu=atoi(argv[12]);
    }else{
        std::cout<<"Numero de parametros erroneos, programa finalizado"<<std::endl;
        std::cout<<"Los parametros son:"<<std::endl;
        std::cout<<"Nx, Ny, M, U, tU,IndexFlux ,p ,L, W ,E0 ,MachineName,GPU"<<std::endl;
        return 0;}
	double tempB=52608*nflux/Nx;
	B=ceil(tempB);
	//std::string String = static_cast<ostringstream*>( &(ostringstream() << B) )->str();
	std::ostringstream ss;ss << B;
	std::cout<<"The magnetic Filed B="<<B<<std::endl;

    /***********Declaracion de directorios de datos de salida **************/
	std::string dospath("data/GrapheneLDOS");
	dospath+="Nx";		dospath+=argv[1];
	dospath+="Ny";		dospath+=argv[2];
	dospath+="M";		dospath+=argv[3];
	dospath+="U";		dospath+=argv[4];
	dospath+="tU";		dospath+=argv[5];
	dospath+="B"     ; dospath+=ss.str();
	dospath+="p";		dospath+=argv[7];
	dospath+="L";		dospath+=argv[8];
	dospath+="W";		dospath+=argv[9];
	dospath+="E0";		dospath+=argv[10];
	dospath+=argv[11] ;
	dospath+=".dat"  ;

	std::string sigmaxxpath("data/GrapheneIMPU");
	sigmaxxpath+="Nx"     ; sigmaxxpath+=argv[1];
	sigmaxxpath+="Ny"     ; sigmaxxpath+=argv[2];
	sigmaxxpath+="M"     ; sigmaxxpath+=argv[3];
	sigmaxxpath+="U"     ; sigmaxxpath+=argv[4];
	sigmaxxpath+="tU"    ; sigmaxxpath+=argv[5];
	sigmaxxpath+="B"     ; sigmaxxpath+=ss.str();
	sigmaxxpath+="p"     ; sigmaxxpath+=argv[7];
	sigmaxxpath+="L"     ; sigmaxxpath+=argv[8];
	sigmaxxpath+="W"     ; sigmaxxpath+=argv[9];
	sigmaxxpath+="E0"	 ; sigmaxxpath+=argv[10];
	sigmaxxpath+=argv[11] ;
	sigmaxxpath+=".dat"  ;

    //Select the gpu device(Default value 0)
   hipSetDevice(ngpu);
	D 	=	Nx*Ny;
	DD	=	2*D;
    HCOO H (DD,DD,4*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins
	graphene::lattice(Nx,Ny,H);
    graphene::TSiteDis(Nx,Ny,H,p,U,tU);
	graphene::Magnetic_Field(Nx,Ny,H,nflux);
	RefineSparse(H);
	//print(H);
	FindingExtrema(H,Emin,Emax,400);
//	FindingExtrema(H,Emin,Emax);
	E0min=0.9*Emin;
	E0max=0.9*Emax;
	LDOS(H,Nx,Ny,M,Emin,Emax,E0,W ,L,dospath);


    return 0;
}
/***************************REEScalando el hamiltoniano******************/
