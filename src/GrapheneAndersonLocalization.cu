#include "hip/hip_runtime.h"
#include "utilidades.h"
#include "linear_cuda_utilities.h"
#include "graphene_lattice_cusp.h"
#include <cusp/transpose.h>
#include <iostream>
#include <fstream>
#include "kpm_cusp_updated.h"
#include <sys/time.h>
#include <unistd.h>
#include <sstream>
//#define FCOMPLEX
#define DCOMPLEX
//#define FLOAT
//#define DOUBLE

#ifdef FCOMPLEX
cusp::complex<float> I(0.0f,1.0f);
cusp::complex<float> zero(0.0f,0.0f);
typedef float    FloatType;
typedef cusp::complex<float>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif

#ifdef DCOMPLEX
cusp::complex<double> I(0.0f,1.0f);
cusp::complex<double> zero(0.0f,0.0f);
typedef double    FloatType;
typedef cusp::complex<double>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif


#ifdef FLOAT
typedef float    FloatType;
typedef int      Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<float> I(0.0f,1.0f);
float zero=0;
#endif

#ifdef DOUBLE
typedef double    Scalar;
typedef double    FloatType;
typedef int       Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<double> I(0.0,1.0);
double zero=0;
#endif


int main(int argc, char *argv[])    			//Para simplificar las corridas elegimos N=argv[1], M=argv[2]
{
    
    /***********Declaracion de apuntadores y variables del programa*********/
    int Nx, Ny,D,DD,M, R,ngpu;	//variables de conteo
	FloatType Emin, Emax,alpha;
	FloatType W,Lx,Ly;
	int SEED;
    int pid=getpid(); // get it as per your OS
	timeval t;
	gettimeofday(&t, NULL);
	std::stringstream ssPID;		//create a stringstream
	ssPID << pid;//add number to the stream
    
    /************Definimos los parametros iniciales **************************/
	if(argc>=10+1){
		Nx=atoi(argv[1]);                                        //Tamaño del sistema
		Ny=atoi(argv[2]);                                        //Tamaño del sistema
		M=atoi(argv[3]);
        W= atof(argv[4]);
        R=atoi(argv[5]);
		Lx=atof(argv[6]);
		Ly=atof(argv[7]);
        //      MachineName=argv[8];
        ngpu=atoi(argv[9]);
        SEED=atoi(argv[10]);
    }else{
        std::cout<<"Numero de parametros erroneos, programa finalizado"<<std::endl;
        std::cout<<"Los parametros son:"<<std::endl;
        std::cout<<"Nx, Ny, M, W, R,Lx, Ly  ,MachineName,GPU"<<std::endl;
        return 0;}
	srand(SEED);

    /***********Declaracion de directorios de datos de salida **************/
	std::string dataID("");
	dataID+="Nx";		dataID+=argv[1];
	dataID+="Ny";		dataID+=argv[2];
	dataID+="M";		dataID+=argv[3];
	dataID+="W";		dataID+=argv[4];
	dataID+="R";		dataID+=argv[5];
	dataID+="Lx";		dataID+=argv[6];
	dataID+="Ly";		dataID+=argv[7];
	dataID+=argv[8] ;

	//Creates the file that will hold the Density of states 
	std::string dospath("data/GrapheneDOS");
	dospath.append(dataID);
	dospath+=".dat";

	//Creates the file that will hold the Density of states 
	std::string olddospath("data/GrapheneOLDDOS");
	olddospath.append(dataID);
	olddospath+=".dat";


	//Creates the file that will hold the Longitudinal Conductivity
	std::string spec_sigmaxx_path("data/GrapheneSpectralConductivityXX");
	spec_sigmaxx_path.append(dataID);
	spec_sigmaxx_path+=".dat";

	//Creates the file that will hold the Longitudinal Conductivity
	std::string spec_sigmaxy_path("data/GrapheneSpectralConductivityXY");
	spec_sigmaxy_path.append(dataID);
	spec_sigmaxy_path+=".dat";


	//Creates the file that will hold the Longitudinal Conductivity
	std::string sigmaxxpath("data/GrapheneConductivityXX");
	sigmaxxpath.append(dataID);
	sigmaxxpath+=".dat";

	//Creates the file that will hold the Density of states 
	std::string dosmappath("data/GrapheneDOSMAP");
	dosmappath.append(dataID);
	dosmappath+=".dat";
	//Creates the file that will hold the Longitudinal Conductivity
	std::string sigmaxxmappath("data/GrapheneConductivityXXMAP");
	sigmaxxmappath.append(dataID);
	sigmaxxmappath+=".dat";

	//Creates the file that will hold the Transverse Conductivity
	std::string sigmaxypath("data/GrapheneMagneticConductivityXY");
	sigmaxypath.append(dataID);
	sigmaxypath+=".dat";


	//Select the gpu device(Default value 0)
	std::cout<<"Setting device "<<ngpu<<std::endl;
   	hipSetDevice(ngpu);
	D 	=	Nx*Ny;
	DD	=	2*D;
	HCOO H (DD,DD,4*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins
	HCOO Vx(DD,DD,4*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins	

	graphene::lattice(Nx,Ny,H);
	graphene::Anderson(Nx,Ny,H,W);
	graphene::velocityx(Nx,Ny,H,Vx);
	RefineSparse(H);
	RefineSparse(Vx);
	Emin=-4.15;
	Emax= 4.15;
	alpha=0.9;
	chebyshev::Rescale(H,Emin,Emax,alpha);
	cycletime(-1);
	std::cout<<"Calculating OLD DOS"<<std::endl;
    chebyshev::random::DOS(H,M,R,Emin,Emax,alpha,	olddospath,1001);
	cycletime(-1);
	cycletime(-1);
	std::cout<<"Calculating Spectral Conductivity"<<std::endl;
	//chebyshev::random::SIGMA(H,Vx,Vx,M,R,Emin,Emax,alpha,spec_sigmaxx_path,spec_sigmaxy_path,65341);
	cycletime(-1);
	cycletime(-1);
	std::cout<<"Calculating DOS and ZERO TEMP Conductivity"<<std::endl;
	//chebyshev::random::LinearDosConductivity(H,Vx,M,R,Emin,Emax,alpha,dospath,sigmaxxpath,-3.0,3.0,100);
	cycletime(-1);
	cycletime(-1);
	std::cout<<"Calculating DOS and ZERO TEMP Conductivity maps"<<std::endl;
	chebyshev::realspace::LinearDosConductivity(Nx,Ny,H,Vx,M,Emin,Emax,alpha,dosmappath,sigmaxxmappath,(FloatType)0.01,sqrt(3)*(Lx+0.5*Ly),1.5*Ly);
	cycletime(-1);


    return 0;
}
/***************************REEScalando el hamiltoniano******************/
