#include "utilidades.h"
#include "linear_cuda_utilities.h"
//#include "graphene_cusp.h"
#include "graphene_lattice_cusp.h"
//#include "lattice_cusp.h"
#include <cusp/transpose.h>
#include <iostream>
#include <fstream>
#include "kpm_cusp_updated.h"
//#include "kpm_cusp.h"

#include <sys/time.h>
#include <unistd.h>
#include <sstream>
//#define FCOMPLEX
#define DCOMPLEX
//#define FLOAT
//#define DOUBLE

#ifdef FCOMPLEX
cusp::complex<float> I(0.0f,1.0f);
cusp::complex<float> zero(0.0f,0.0f);
typedef float    FloatType;
typedef cusp::complex<float>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif

#ifdef DCOMPLEX
cusp::complex<double> I(0.0f,1.0f);
cusp::complex<double> zero(0.0f,0.0f);
typedef double    FloatType;
typedef cusp::complex<double>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif


#ifdef FLOAT
typedef float    FloatType;
typedef int      Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<float> I(0.0f,1.0f);
float zero=0;
#endif

#ifdef DOUBLE
typedef double    Scalar;
typedef double    FloatType;
typedef int       Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<double> I(0.0,1.0);
double zero=0;
#endif


int main(int argc, char *argv[])    			//Para simplificar las corridas elegimos N=argv[1], M=argv[2]
{
    /***********Declaracion de apuntadores y variables del programa*********/
    int Nx, Ny, D, DD, M, R,ngpu;	//variables de conteo
	FloatType Emin, Emax,alpha;
	FloatType U,V0,p,eps0;
	int SEED;
    int pid=getpid(); // get it as per your OS
	timeval t;
	gettimeofday(&t, NULL);
	std::stringstream ssPID;		//create a stringstream
	ssPID << pid;//add number to the stream
    
    /************Definimos los parametros iniciales **************************/
	if(argc>=10+1){
		Nx=atoi(argv[1]);                                        //Tamaño del sistema
		Ny=atoi(argv[2]);                                        //Tamaño del sistema
		M =atoi(argv[3]);
        U = atof(argv[4]);
        V0= atof(argv[5]);
        p = atof(argv[6]);
        R = atof(argv[7]);
        //      MachineName=argv[8];
        ngpu=atoi(argv[9]);
        SEED=atoi(argv[10]);
		}else{
        std::cout<<"Numero de parametros erroneos, programa finalizado"<<std::endl;
        std::cout<<"Los parametros son:"<<std::endl;
        std::cout<<"Nx, Ny, M, U, V0, p, R, MachineName,GPU, SEED"<<std::endl;
        return 0;}
	srand(SEED);

    /***********Declaracion de directorios de datos de salida **************/
	std::string dataID("");
	dataID+="Nx";		dataID+=argv[1];
	dataID+="Ny";		dataID+=argv[2];
	dataID+="M";		dataID+=argv[3];
	dataID+="U";		dataID+=argv[4];
	dataID+="V0"     ;   dataID+=argv[5];
	dataID+="p"     ;   dataID+=argv[6];
	dataID+="R";		dataID+=argv[7];
	dataID+=argv[8] ;

	//Creates the file that will hold the Density of states 
	std::string dospath("data/GrapheneHsiteImpDOS");
	dospath.append(dataID);
	dospath+=".dat";
	//Creates the file that will hold the Longitudinal Conductivity
	std::string sigmaxxpath("data/GrapheneHsiteImpConductivityXX");
	sigmaxxpath.append(dataID);
	sigmaxxpath+=".dat";
	//Creates the file that will hold the Transverse Conductivity
	std::string sigmaxypath("data/GrapheneHsiteImpConductivityXY");
	sigmaxypath.append(dataID);
	sigmaxypath+=".dat";


	//Select the gpu device(Default value 0)
	std::cout<<"Setting device "<<ngpu<<std::endl;
   	hipSetDevice(ngpu);
/*As shown in the following draw
*	    V1=(i,j+1)_B ____ V2=(i,j+1)_A
*			/    \
*	    V0=(i,j)_A /      \V3=(i+1,j+1)_B
*		       \      /
*	    V5=(i+1,j)_B\____/V4=(i+1,j)_A
*
* the vector V[6]=(V0,V1,V2,V3,V4,V5) has as components the hybridization amplitude
* of the atoms in the ring within the Anderson Impurity model
*/ 
	//Stype
	Scalar V[]={V0,V0,V0,V0,V0,V0}; eps0=1; Emin=-4.0;Emax= 4.0;
	//Ftype
//	Scalar V[]={V0,-V0,V0,-V0,V0,-V0}; Emin=-4.6;Emax= 4.6; eps0=-1;
	//Ptype
//	Scalar V[]={V0,0,0,-V0,0,0};
	//D1type
//	Scalar V[]={V0,-0.5*V0,-0.5*V0,V0,-0.5*V0,-0.5*V0};
	//D2type
//	Scalar V[]={0,V0,-V0,0,V0,-V0};

	D 	=	Nx*Ny;
	DD	=	2*D;
	HCOO H (DD,DD,13*DD); // 13 vecinos=1onsite+3primeros+6segundos+3terceros
	HCOO Vx(DD,DD,13*DD); // 13 vecinos=1onsite+3primeros+6segundos+3terceros	
	HCOO Vy(DD,DD,13*DD); // 13 vecinos=1onsite+3primeros+6segundos+3terceros
	graphene::lattice(Nx,Ny,H);
	graphene::impurities::HtypeDistribution(Nx,Ny,H,p,(double)0,V,eps0);
//	graphene::Anderson(Nx,Ny,H,U);
	graphene::velocityx(Nx,Ny,H,Vx);
	graphene::velocityy(Nx,Ny,H,Vy);
	linalg::SpectralBounds(H,Emin,Emax,0.001);
	RefineSparse(H);
	RefineSparse(Vx);
	RefineSparse(Vy);
	alpha=0.9;
	chebyshev::Rescale(H,Emin,Emax,alpha);
	std::cout<<std::endl<<"Calculating DOS"<<std::endl;
	srand(time(0)*t.tv_usec * t.tv_sec * pid);
	cycletime(-1);
	chebyshev::random::DOS(H,M,R,Emin,Emax,alpha,dospath,65536+1);
	cycletime(-1);
	std::cout<<"Calculating SIGMAXX and SIGMAXY BASTIN"<<std::endl;
	chebyshev::random::SIGMA(H,Vx,Vy,M,R,Emin,Emax,alpha,sigmaxxpath,sigmaxypath,65536+1);


    return 0;
}
/***************************REEScalando el hamiltoniano******************/
