#include "utilidades.h"
#include "linear_cuda_utilities.h"
//#include "graphene_cusp.h"
#include "lattice_cusp.h"
#include <cusp/transpose.h>
#include <iostream>
#include <fstream>
#include "kpm_cusp.h"
#include <sys/time.h>
#include <unistd.h>
#include <sstream>
//#define FCOMPLEX
#define DCOMPLEX
//#define FLOAT
//#define DOUBLE

#ifdef FCOMPLEX
cusp::complex<float> I(0.0f,1.0f);
cusp::complex<float> zero(0.0f,0.0f);
typedef float    FloatType;
typedef cusp::complex<float>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif

#ifdef DCOMPLEX
cusp::complex<double> I(0.0f,1.0f);
cusp::complex<double> zero(0.0f,0.0f);
typedef double    FloatType;
typedef cusp::complex<double>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif


#ifdef FLOAT
typedef float    FloatType;
typedef int      Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<float> I(0.0f,1.0f);
float zero=0;
#endif

#ifdef DOUBLE
typedef double    Scalar;
typedef double    FloatType;
typedef int       Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<double> I(0.0,1.0);
double zero=0;
#endif


int main(int argc, char *argv[])    			//Para simplificar las corridas elegimos N=argv[1], M=argv[2]
{
    
    /***********Declaracion de apuntadores y variables del programa*********/
    int Nx, Ny,D,DD,M, R,ngpu;	//variables de conteo
	FloatType Emin, Emax,alpha;
	FloatType W;
	int SEED;
    int pid=getpid(); // get it as per your OS
	timeval t;
	gettimeofday(&t, NULL);
	std::stringstream ssPID;		//create a stringstream
	ssPID << pid;//add number to the stream
    
    /************Definimos los parametros iniciales **************************/
	if(argc>=8+1){
		Nx=atoi(argv[1]);                                        //Tamaño del sistema
		Ny=atoi(argv[2]);                                        //Tamaño del sistema
		M=atoi(argv[3]);
		W=atof(argv[4]);
		R=atoi(argv[5]);
        //      MachineName=argv[6];
        ngpu=atoi(argv[7]);
        SEED=atoi(argv[8]);
    }else{
        std::cout<<"Numero de parametros erroneos, programa finalizado"<<std::endl;
        std::cout<<"Los parametros son:"<<std::endl;
        std::cout<<"Nx, Ny, M, W,R  ,MachineName,GPU"<<std::endl;
        return 0;}
	srand(SEED);

    /***********Declaracion de directorios de datos de salida **************/
	std::string dataID("");
	dataID+="Nx";		dataID+=argv[1];
	dataID+="Ny";		dataID+=argv[2];
	dataID+="M";		dataID+=argv[3];
	dataID+="W";		dataID+=argv[4];
	dataID+="R";		dataID+=argv[5];
	dataID+=argv[6] ;

	//Creates the file that will hold the Density of states 
	std::string dospath("data/GrapheneAndersonDOS");
	dospath.append(dataID);
	dospath+=".dat";
	//Creates the file that will hold the Longitudinal Conductivity
	std::string sigmaxxpath("data/GrapheneAndersonConductivityXX");
	sigmaxxpath.append(dataID);
	sigmaxxpath+=".dat";
	//Creates the file that will hold the Transverse Conductivity
	std::string sigmaxypath("data/GrapheneAndersonConductivityXY");
	sigmaxypath.append(dataID);
	sigmaxypath+=".dat";


	//Select the gpu device(Default value 0)
	std::cout<<"Setting device "<<ngpu<<std::endl;
   	hipSetDevice(ngpu);
	D 	=	Nx*Ny;
	DD	=	2*D;
	HCOO H (DD,DD,4*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins
	HCOO Vx(DD,DD,4*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins	
	HCOO Vy(DD,DD,4*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins	

	graphene::lattice(Nx,Ny,H);
	graphene::Anderson(Nx,Ny,H,W);
	graphene::velocityx(Nx,Ny,H,Vx);
	graphene::velocityy(Nx,Ny,H,Vy);
	RefineSparse(H);
	RefineSparse(Vx);
	RefineSparse(Vy);
	Emin=-3.05-0.5*W;
	Emax= 3.05+0.5*W;
	alpha=0.1;
	chebyshev::Rescale(H,Emin,Emax,alpha);
	cycletime(-1);
	std::cout<<std::endl<<"Calculating DOS"<<std::endl;
	srand(time(0)*t.tv_usec * t.tv_sec * pid);
	chebyshev::random::DOS(H,M,R,Emin,Emax,alpha,dospath,65536+1);
	cycletime(-1);
	std::cout<<"Calculating SIGMAXX and SIGMAXY BASTIN"<<std::endl;
//	chebyshev::random::SIGMA(H,Vx,Vy,M,R,Emin,Emax,alpha,sigmaxxpath,sigmaxypath,65536+1);


    return 0;
}
/***************************REEScalando el hamiltoniano******************/
