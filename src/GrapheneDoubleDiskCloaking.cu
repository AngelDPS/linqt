#include "hip/hip_runtime.h"
#include "utilidades.h"
#include "linear_cuda_utilities.h"
//#include "graphene_cusp.h"
#include "graphene_lattice_cusp.h"
//#include "lattice_cusp.h"
#include <cusp/transpose.h>
#include <iostream>
#include <fstream>
#include "kpm_cusp_updated.h"
//#include "kpm_cusp.h"

#include <sys/time.h>
#include <unistd.h>
#include <sstream>
//#define FCOMPLEX
#define DCOMPLEX
//#define FLOAT
//#define DOUBLE

#ifdef FCOMPLEX
cusp::complex<float> I(0.0f,1.0f);
cusp::complex<float> zero(0.0f,0.0f);
typedef float    FloatType;
typedef cusp::complex<float>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif

#ifdef DCOMPLEX
cusp::complex<double> I(0.0f,1.0f);
cusp::complex<double> zero(0.0f,0.0f);
typedef double    FloatType;
typedef cusp::complex<double>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif


#ifdef FLOAT
typedef float    FloatType;
typedef int      Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<float> I(0.0f,1.0f);
float zero=0;
#endif

#ifdef DOUBLE
typedef double    Scalar;
typedef double    FloatType;
typedef int       Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<double> I(0.0,1.0);
double zero=0;
#endif


int main(int argc, char *argv[])    			//Para simplificar las corridas elegimos N=argv[1], M=argv[2]
{
    /***********Declaracion de apuntadores y variables del programa*********/
    int Nx, Ny, D, DD, M, R,ngpu;	//variables de conteo
	FloatType Emin, Emax,alpha;
	FloatType W, Umin, Umax, Rmin, Rmax;
	int SEED;
    int pid=getpid(); // get it as per your OS
	timeval t;
	gettimeofday(&t, NULL);
	std::stringstream ssPID;		//create a stringstream
	ssPID << pid;//add number to the stream
    
    /************Definimos los parametros iniciales **************************/
	if(argc>=12+1){
		Nx	=atoi(argv[1]);                                        //Tamaño del sistema
		Ny	=atoi(argv[2]);                                        //Tamaño del sistema
		M 	=atoi(argv[3]);
        W 	=atof(argv[4]);
        Umin=atof(argv[5]);
        Rmin=atof(argv[6]);
        Umax=atof(argv[7]);
        Rmax=atof(argv[8]);
        R = atof(argv[9]);
        //      MachineName=argv[10];
        ngpu=atoi(argv[11]);
        SEED=atoi(argv[12]);
		}else{
        std::cout<<"Numero de parametros erroneos, programa finalizado"<<std::endl;
        std::cout<<"Los parametros son:"<<std::endl;
        std::cout<<"Nx, Ny, M, W, Umin, Rmin, Umax, Rmax, R, MachineName,GPU, SEED"<<std::endl;
        return 0;}
	srand(SEED);

    /***********Declaracion de directorios de datos de salida **************/
	std::string dataID("");
	dataID+="Nx";		dataID+=argv[1];
	dataID+="Ny";		dataID+=argv[2];
	dataID+="M";		dataID+=argv[3];
	dataID+="W";		dataID+=argv[4];
	dataID+="Umin";		dataID+=argv[5];
	dataID+="Rmin";		dataID+=argv[6];
	dataID+="Umax";		dataID+=argv[7];
	dataID+="Rmax";		dataID+=argv[8];
	dataID+="R";		dataID+=argv[9];
	dataID+=argv[8] ;

	//Creates the file that will hold the Density of states 
	std::string dospath("data/GrapheneDoubleDiskDOS");
	dospath.append(dataID);
	dospath+=".dat";
	//Creates the file that will hold the Longitudinal Conductivity
	std::string sigmaxxpath("data/GrapheneDoubleDiskCondXX");
	sigmaxxpath.append(dataID);
	sigmaxxpath+=".dat";
	//Creates the file that will hold the Density of states 
	std::string dosmappath("data/GrapheneDoubleDiskDOSMAP");
	dosmappath.append(dataID);
	dosmappath+=".dat";
	//Creates the file that will hold the Longitudinal Conductivity
	std::string sigmaxxmappath("data/GrapheneDoubleDiskCondXXMAP");
	sigmaxxmappath.append(dataID);
	sigmaxxmappath+=".dat";
	//Select the gpu device(Default value 0)
	std::cout<<"Setting device "<<ngpu<<std::endl;
   	hipSetDevice(ngpu);


	D 	=	Nx*Ny;
	DD	=	2*D;
	HCOO H (DD,DD,4*DD); // 13 vecinos=1onsite+3primeros+6segundos+3terceros
	HCOO Vx(DD,DD,4*DD); // 13 vecinos=1onsite+3primeros+6segundos+3terceros	

	graphene::lattice(Nx,Ny,H);
	graphene::Anderson(Nx,Ny,H,W);
	graphene::potentials::CenteredDoubleDisk(Nx,Ny,H,Umin,Rmin,Umax,Rmax);
	graphene::velocityx(Nx,Ny,H,Vx);
	Emin=-3.13; Emax= 3.13;
	linalg::SpectralBounds(H,Emin,Emax,0.001);
	RefineSparse(H);
	RefineSparse(Vx);
	alpha=1.0;
	chebyshev::Rescale(H,Emin,Emax,alpha);
	std::cout<<std::endl<<"Calculating DOS"<<std::endl;
	srand(time(0)*t.tv_usec * t.tv_sec * pid);
	cycletime(-1);
	chebyshev::random::DOS(H,M,R,Emin,Emax,alpha,dospath,1000+1);
	chebyshev::realspace::LinearDosConductivity(Nx,Ny,H,Vx,M,Emin,Emax,dosmappath,sigmaxxmappath,(FloatType)0.01,sqrt(3)*(Nx+0.5*Ny),1.5*Ny);
//	cycletime(-1);
//	std::cout<<"Calculating SIGMAXX and SIGMAXY BASTIN"<<std::endl;
//	chebyshev::random::SIGMA(H,Vx,Vy,M,R,Emin,Emax,alpha,sigmaxxpath,sigmaxypath,65536+1);


    return 0;
}
/***************************REEScalando el hamiltoniano******************/
