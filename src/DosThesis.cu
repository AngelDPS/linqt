#include "utilidades.h"
#include "linear_cuda_utilities.h"
//#include "graphene_cusp.h"
#include "lattice_cusp.h"
#include <cusp/transpose.h>
#include <iostream>
#include <fstream>
#include "kpm_cusp.h"
#include <sys/time.h>
#include <unistd.h>
#include <sstream>
//#define FCOMPLEX
#define DCOMPLEX
//#define FLOAT
//#define DOUBLE

#ifdef FCOMPLEX
cusp::complex<float> I(0.0f,1.0f);
cusp::complex<float> zero(0.0f,0.0f);
typedef float    FloatType;
typedef cusp::complex<float>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif

#ifdef DCOMPLEX
cusp::complex<double> I(0.0f,1.0f);
cusp::complex<double> zero(0.0f,0.0f);
typedef double    FloatType;
typedef cusp::complex<double>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif


#ifdef FLOAT
typedef float    FloatType;
typedef int      Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<float> I(0.0f,1.0f);
float zero=0;
#endif

#ifdef DOUBLE
typedef double    Scalar;
typedef double    FloatType;
typedef int       Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<double> I(0.0,1.0);
double zero=0;
#endif


int main(int argc, char *argv[])    			//Para simplificar las corridas elegimos N=argv[1], M=argv[2]
{
    
    /***********Declaracion de apuntadores y variables del programa*********/
    int Nx, Ny,D,DD,M, R;	//variables de conteo
	FloatType Emin, Emax,alpha;
	FloatType U;
	int ngpu;
	int SEED;
    int pid=getpid(); // get it as per your OS
	timeval t;
	gettimeofday(&t, NULL);
	std::stringstream ssPID;		//create a stringstream
	ssPID << pid;//add number to the stream
    
    /************Definimos los parametros iniciales **************************/
	if(argc>=8+1){
		Nx=atoi(argv[1]);                                        //Tamaño del sistema
		Ny=atoi(argv[2]);                                        //Tamaño del sistema
		M=atoi(argv[3]);
        U= atof(argv[4]);
        R=atoi(argv[5]);
        //      MachineName=argv[6];
        ngpu=atoi(argv[7]);
        SEED=atoi(argv[8]);
    }else{
        std::cout<<"Numero de parametros erroneos, programa finalizado"<<std::endl;
        std::cout<<"Los parametros son:"<<std::endl;
        std::cout<<"Nx, Ny, M, U ,R  ,MachineName,GPU"<<std::endl;
        return 0;}
	srand(SEED);

    /***********Declaracion de directorios de datos de salida **************/
	std::string dataID("");
	dataID+="Nx";		dataID+=argv[1];
	dataID+="Ny";		dataID+=argv[2];
	dataID+="M";		dataID+=argv[3];
	dataID+="U";		dataID+=argv[4];
	dataID+="R";		dataID+=argv[5];
	dataID+=argv[6] ;

	//Creates the file that will hold the Density of states 
	std::string dospath("data/GrapheneDOS");
	dospath.append(dataID);
	dospath+=".dat";

	//Select the gpu device(Default value 0)
	std::cout<<"Setting device "<<ngpu<<std::endl;
   	hipSetDevice(ngpu);
	D 	=	Nx*Ny;
	DD	=	2*D;
	HCOO H (DD,DD,4*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins

	graphene::lattice(Nx,Ny,H);
	graphene::Anderson(Nx,Ny,H,U);
	RefineSparse(H);
	Emin=-3.1;
	Emax= 3.1;
	alpha=0.9;
	chebyshev::Rescale(H,Emin,Emax,alpha);
	cycletime(-1);
	std::cout<<std::endl<<"Calculating DOS"<<std::endl;
	srand(time(0)*t.tv_usec * t.tv_sec * pid);
	chebyshev::random::DOS(H,M,R,Emin,Emax,alpha,dospath,6556+1);
	cycletime(-1);


    return 0;
}
/***************************REEScalando el hamiltoniano******************/
