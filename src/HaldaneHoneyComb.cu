#include "utilidades.h"
#include "linear_cuda_utilities.h"
//#include "graphene_cusp.h"
#include "lattice_cusp.h"
#include <iostream>
#include <fstream>
#include "kpm_cusp.h"
#include <sys/time.h>
#include <unistd.h>
#include <sstream>

#define FCOMPLEX
//#define DCOMPLEX
//#define FLOAT
//#define DOUBLE

#ifdef FCOMPLEX
cusp::complex<float> I(0.0f,1.0f);
cusp::complex<float> zero(0.0f,0.0f);
typedef float    FloatType;
typedef cusp::complex<float>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif

#ifdef DCOMPLEX
cusp::complex<double> I(0.0f,1.0f);
cusp::complex<double> zero(0.0f,0.0f);
typedef double    FloatType;
typedef cusp::complex<double>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif


#ifdef FLOAT
typedef float    Scalar	HCOO Vx(DDD,DDD,S*4*DDD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins	
;
typedef float    FloatType;
typedef int      Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<float> I(0.0f,1.0f);
float zero=0;
#endif

#ifdef DOUBLE
typedef double    Scalar;
typedef double    FloatType;
typedef int       Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<double> I(0.0,1.0);
double zero=0;
#endif


int main(int argc, char *argv[])    			//Para simplificar las corridas elegimos N=argv[1], M=argv[2]
{
    
    /***********Declaracion de apuntadores y variables del programa*********/
    int Nx, Ny,D,DD,M, R,ngpu,En,seed;	//variables de conteo
	FloatType   Emin, Emax,E0min, E0max;
	FloatType U,UAB,p,lambda,alpha;
    int pid=getpid(); // get it as per your OS
	timeval t;
	gettimeofday(&t, NULL);
	std::stringstream ssPID;		//create a stringstream
	ssPID << pid;//add number to the stream
    /************Definimos los parametros iniciales **************************/
	if(argc>=13+1){
		Nx=atoi(argv[1]);                                        //Tamaño del sistema
		Ny=atoi(argv[2]);                                        //Tamaño del sistema
		M=atoi(argv[3]);
        U= atof(argv[4]);
		lambda=atof(argv[5]);
		UAB=atof(argv[6]);
		p=atof(argv[7]);                                     //Densidad de impurezas
        R=atoi(argv[8]);
		alpha=atof(argv[9]);                                     //Densidad de impurezas
        En=atoi(argv[10]);
        //MachineName=argv[11];
        ngpu=atoi(argv[12]);
	seed=atoi(argv[13]);
		}else{
        std::cout<<"Numero de parametros erroneos, programa finalizado"<<std::endl;
        std::cout<<"Los parametros son:"<<std::endl;
        std::cout<<"Nx, Ny, M, U,lambda, UAB ,p ,R,alpha,En  ,MachineName,GPU, seed"<<std::endl;
        return 0;}
	srand(seed);

    /***********Declaracion de directorios de datos de salida **************/
	std::string dataID("");
	dataID+="Nx";		dataID+=argv[1];
	dataID+="Ny";		dataID+=argv[2];
	dataID+="M";		dataID+=argv[3];
	dataID+="U";		dataID+=argv[4];
	dataID+="lambda";	dataID+=argv[5];
	dataID+="UAB";		dataID+=argv[6];
	dataID+="p";		dataID+=argv[7];
	dataID+="R";		dataID+=argv[8];
	dataID+="alpha";		dataID+=argv[9];
	dataID+="En";		dataID+=argv[10];
		dataID+=argv[11] ;


	//Creates the file that will hold oll the stdout information 
	//Creates the file that will hold the Density of states 
	std::string dospath("data/GrapheneHaldaneDOS");
	dospath.append(dataID);
	dospath+=".dat";
	//Creates the file that will hold the Longitudinal Conductivity
	std::string sigmaxxpath("data/GrapheneHaldaneConductivityXX");
	sigmaxxpath.append(dataID);
	sigmaxxpath+=".dat";
	//Creates the file that will hold the Transverse Conductivity
	std::string sigmaxypath("data/GrapheneHaldaneConductivityXY");
	sigmaxypath.append(dataID);
	sigmaxypath+=".dat";

	

    //Select the gpu device(Default value 0)
   hipSetDevice(ngpu);
	D 	=	Nx*Ny;
	DD	=	2*D;
    	HCOO H (DD,DD,4*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins
	HCOO Vx(DD,DD,4*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins	
	HCOO Vy(DD,DD,4*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins	

	graphene::lattice(Nx,Ny,H);
//    graphene::LatticePotential(Nx,Ny,H,(double)1,UAB);
//	graphene::TSiteDis(Nx,Ny,H,p,U,(FloatType)0);
//	graphene::Anderson(Nx,Ny,H,U);
//	graphene::HaldaneHoneyComb(Nx,Ny,H,lambda);
	graphene::velocityx(Nx,Ny,H,Vx);
	graphene::velocityy(Nx,Ny,H,Vy);
	RefineSparse(H);
	RefineSparse(Vx);
	RefineSparse(Vy);
	E0min=-En*3.45;
	E0max= En*3.45;
	Emin= E0min;
	Emax= E0max;
	chebyshev::Rescale(H,Emin,Emax,alpha);

	srand(time(0)*t.tv_usec * t.tv_sec * pid);
 	std::cout<<std::endl<<"Calculating DOS"<<std::endl;
	cycletime(-1);
//	chebyshev::random::DOS(H,M,R,Emin,Emax,alpha,dospath,	16384); //falta un 8 al final
	cycletime(-1);
	std::cout<<"Calculating SIGMAXX and SIGMAXY BASTIN"<<std::endl;
	chebyshev::random::SIGMA(H,Vx,Vy,M,R,Emin,Emax,alpha,sigmaxxpath,sigmaxypath,65536+1);
	cycletime(-1);

    return 0;
}
/***************************REEScalando el hamiltoniano******************/
	
