#include "hip/hip_runtime.h"
#include "utilidades.h"
#include "linear_cuda_utilities.h"
//#include "graphene_cusp.h"
#include "lattice_cusp.h"
#include <cusp/transpose.h>
#include <iostream>
#include <fstream>
#include "kpm_cusp.h"
#include <sys/time.h>
#include <unistd.h>
#include <sstream>
//#define FCOMPLEX
#define DCOMPLEX
//#define FLOAT
//#define DOUBLE

#ifdef FCOMPLEX
cusp::complex<float> I(0.0f,1.0f);
cusp::complex<float> zero(0.0f,0.0f);
typedef float    FloatType;
typedef cusp::complex<float>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif

#ifdef DCOMPLEX
cusp::complex<double> I(0.0f,1.0f);
cusp::complex<double> zero(0.0f,0.0f);
typedef double    FloatType;
typedef cusp::complex<double>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif


#ifdef FLOAT
typedef float    FloatType;
typedef int      Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<float> I(0.0f,1.0f);
float zero=0;
#endif

#ifdef DOUBLE
typedef double    Scalar;
typedef double    FloatType;
typedef int       Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<double> I(0.0,1.0);
double zero=0;
#endif


int main(int argc, char *argv[])    			//Para simplificar las corridas elegimos N=argv[1], M=argv[2]
{
    
    /***********Declaracion de apuntadores y variables del programa*********/
    int Nx, Ny,D,DD,M, R,ngpu,nflux,B;	//variables de conteo
	FloatType Emin, Emax,alpha;
	FloatType U;
	int SEED;
    int pid=getpid(); // get it as per your OS
	timeval t;
	gettimeofday(&t, NULL);
	std::stringstream ssPID;		//create a stringstream
	ssPID << pid;//add number to the stream
    
    /************Definimos los parametros iniciales **************************/
	if(argc>=9+1){
		Nx=atoi(argv[1]);                                        //Tamaño del sistema
		Ny=atoi(argv[2]);                                        //Tamaño del sistema
		M=atoi(argv[3]);
        U= atof(argv[4]);
		nflux  =atoi(argv[5]);
        R=atoi(argv[6]);
        //      MachineName=argv[7];
        ngpu=atoi(argv[8]);
        SEED=atoi(argv[9]);
    }else{
        std::cout<<"Numero de parametros erroneos, programa finalizado"<<std::endl;
        std::cout<<"Los parametros son:"<<std::endl;
        std::cout<<"Nx, Ny, M, U,IndexFlux ,R  ,MachineName,GPU"<<std::endl;
        return 0;}
	srand(SEED);
	double tempB=157831*nflux/Ny;
	B=ceil(tempB);
	std::string String = static_cast<std::ostringstream*>( &(std::ostringstream() << B) )->str();
	std::ostringstream ss;ss << B;
	std::cout<<"The magnetic Filed B="<<B<<std::endl;

    /***********Declaracion de directorios de datos de salida **************/
	std::string dataID("");
	dataID+="Nx";		dataID+=argv[1];
	dataID+="Ny";		dataID+=argv[2];
	dataID+="M";		dataID+=argv[3];
	dataID+="U";		dataID+=argv[4];
	dataID+="B"     ;   dataID+=ss.str();
	dataID+="R";		dataID+=argv[6];
	dataID+=argv[7] ;

	//Creates the file that will hold the Density of states 
	std::string dospath("data/GrapheneMagneticDOS");
	dospath.append(dataID);
	dospath+=".dat";
	//Creates the file that will hold the Longitudinal Conductivity
	std::string sigmaxxpath("data/GrapheneMagneticConductivityXX");
	sigmaxxpath.append(dataID);
	sigmaxxpath+=".dat";
	//Creates the file that will hold the Transverse Conductivity
	std::string sigmaxypath("data/GrapheneMagneticConductivityXY");
	sigmaxypath.append(dataID);
	sigmaxypath+=".dat";


	//Select the gpu device(Default value 0)
	std::cout<<"Setting device "<<ngpu<<std::endl;
   	hipSetDevice(ngpu);
	D 	=	Nx*Ny;
	DD	=	2*D;
	HCOO H (DD,DD,4*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins
	HCOO Vx(DD,DD,4*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins	
	HCOO Vy(DD,DD,4*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins	

	graphene::lattice(Nx,Ny,H);
	graphene::Anderson(Nx,Ny,H,U);
	graphene::Magnetic_Field(Nx,Ny,H,nflux);
	graphene::velocityx(Nx,Ny,H,Vx);
	graphene::velocityy(Nx,Ny,H,Vy);
	RefineSparse(H);
	RefineSparse(Vx);
	RefineSparse(Vy);
	Emin=-3.5;
	Emax= 3.5;
	alpha=0.9;
	chebyshev::Rescale(H,Emin,Emax,alpha);

	std::ofstream outputFilename0 ("graphene+pristine.Ham.txt");
	std::ofstream outputFilename1 ("graphene+pristine.Vx.txt");
	std::ofstream outputFilename2 ("graphene+pristine.Vy.txt");

	for(int n=0;n<H.num_entries;n++)
		outputFilename0<<H.row_indices[n] <<" "<<H.column_indices[n]<<" "  << H.values[n].real()<<" " <<H.values[n].imag()<<std::endl;

	for(int n=0;n<Vx.num_entries;n++)
		outputFilename1<<Vx.row_indices[n]<<" "<<Vx.column_indices[n]<<" "<< Vx.values[n].real()<<" "<<Vx.values[n].imag()<<std::endl;


	for(int n=0;n<Vy.num_entries;n++)
		outputFilename2<<Vy.row_indices[n]<<" "<<Vy.column_indices[n]<<" "<< Vy.values[n].real()<<" "<<Vy.values[n].imag()<<std::endl;

	outputFilename0.close();
	outputFilename1.close();
	outputFilename2.close();
//		cycletime(-1);
//	std::cout<<std::endl<<"Calculating DOS"<<std::endl;
//	srand(time(0)*t.tv_usec * t.tv_sec * pid);
	chebyshev::random::DOS(H,M,R,Emin,Emax,alpha,dospath,65536+1);
//	cycletime(-1);
//	std::cout<<"Calculating SIGMAXX and SIGMAXY BASTIN"<<std::endl;
	chebyshev::random::SIGMA(H,Vx,Vy,M,R,Emin,Emax,alpha,sigmaxxpath,sigmaxypath,65536+1);

    return 0;
}
/***************************REEScalando el hamiltoniano******************/
