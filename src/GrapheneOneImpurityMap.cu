#include "utilidades.h"
#include "linear_cuda_utilities.h"
//#include "graphene_cusp.h"
#include "graphene_lattice_cusp.h"
//#include "lattice_cusp.h"
#include <cusp/transpose.h>
#include <iostream>
#include <fstream>
#include "kpm_cusp_updated.h"
//#include "kpm_cusp.h"

#include <sys/time.h>
#include <unistd.h>
#include <sstream>
//#define FCOMPLEX
#define DCOMPLEX
//#define FLOAT
//#define DOUBLE

#ifdef FCOMPLEX
cusp::complex<float> I(0.0f,1.0f);
cusp::complex<float> zero(0.0f,0.0f);
typedef float    FloatType;
typedef cusp::complex<float>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif

#ifdef DCOMPLEX
cusp::complex<double> I(0.0f,1.0f);
cusp::complex<double> zero(0.0f,0.0f);
typedef double    FloatType;
typedef cusp::complex<double>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif


#ifdef FLOAT
typedef float    FloatType;
typedef int      Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<float> I(0.0f,1.0f);
float zero=0;
#endif

#ifdef DOUBLE
typedef double    Scalar;
typedef double    FloatType;
typedef int       Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<double> I(0.0,1.0);
double zero=0;
#endif


int main(int argc, char *argv[])    			//Para simplificar las corridas elegimos N=argv[1], M=argv[2]
{
    /***********Declaracion de apuntadores y variables del programa*********/
    int Nx, Ny, D, DD, M,ngpu;	//variables de conteo
	FloatType Emin, Emax;
	FloatType U,V0,eps0;
	int SEED;
    int pid=getpid(); // get it as per your OS
	timeval t;
	gettimeofday(&t, NULL);
	std::stringstream ssPID;		//create a stringstream
	ssPID << pid;//add number to the stream
    
    /************Definimos los parametros iniciales **************************/
	if(argc>=8+1){
		Nx=atoi(argv[1]);                                        //Tamaño del sistema
		Ny=atoi(argv[2]);                                        //Tamaño del sistema
		M =atoi(argv[3]);
        U = atof(argv[4]);
        V0= atof(argv[5]);
        //      MachineName=argv[6];
        ngpu=atoi(argv[7]);
        SEED=atoi(argv[8]);
		}else{
        std::cout<<"Numero de parametros erroneos, programa finalizado"<<std::endl;
        std::cout<<"Los parametros son:"<<std::endl;
        std::cout<<"Nx, Ny, M, U, V0, MachineName,GPU, SEED"<<std::endl;
        return 0;}
	srand(SEED);

    /***********Declaracion de directorios de datos de salida **************/
	std::string dataID("");
	dataID+="Nx";		dataID+=argv[1];
	dataID+="Ny";		dataID+=argv[2];
	dataID+="M";		dataID+=argv[3];
	dataID+="U";		dataID+=argv[4];
	dataID+="V0"     ;   dataID+=argv[5];
	dataID+=argv[6] ;

	std::string dosmappath("data/GrapheneHsiteOneImpDOS");
	dosmappath.append(dataID);
	dosmappath+=".dat";
	//Creates the file that will hold the Longitudinal Conductivity
	std::string condmappath("data/GrapheneHsiteOneImpCond");
	condmappath.append(dataID);
	condmappath+=".dat";

	//Select the gpu device(Default value 0)
	std::cout<<"Setting device "<<ngpu<<std::endl;
   	hipSetDevice(ngpu);


	Scalar V[]={V0,V0,V0,V0,V0,V0}; eps0=1; 
	D 	=	Nx*Ny;
	DD	=	2*D;
	HCOO H (DD,DD,13*DD); // 13 vecinos=1onsite+3primeros+6segundos+3terceros
	HCOO Vx(DD,DD,13*DD); // 13 vecinos=1onsite+3primeros+6segundos+3terceros	
	graphene::lattice(Nx,Ny,H);
	graphene::impurities::Htype(Nx,Ny,H,(int)(Nx/2.0),(int)(Ny/2.0),U,V,eps0);
	graphene::velocityx(Nx,Ny,H,Vx);
	Emin=-4.5;
	Emax= 4.5;
	RefineSparse(H);
	RefineSparse(Vx);
//	linalg::SpectralBounds(H,Emin,Emax,0.0001);
	//print(H);
	chebyshev::Rescale(H,Emin,Emax,1.0);
//	chebyshev::random::DOS(H,M,10,Emin,Emax,(double)0.9,dosmappath,1000);
	chebyshev::realspace::LinearDosConductivity(Nx,Ny,H,Vx,M,Emin,Emax,dosmappath,condmappath,(FloatType)0.01,10.0,10.0);


    return 0;
}
/***************************REEScalando el hamiltoniano******************/
