#include "hip/hip_runtime.h"
#include "utilidades.h"
#include "linear_cuda_utilities.h"
//#include "graphene_cusp.h"
#include "lattice_cusp.h"
#include <cusp/transpose.h>
#include <iostream>
#include <fstream>
#include "kpm_cusp.h"
#include <sys/time.h>
#include <unistd.h>
#include <sstream>
//#define FCOMPLEX
#define DCOMPLEX
//#define FLOAT
//#define DOUBLE

#ifdef FCOMPLEX
cusp::complex<float> I(0.0f,1.0f);
cusp::complex<float> zero(0.0f,0.0f);
typedef float    FloatType;
typedef cusp::complex<float>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif

#ifdef DCOMPLEX
cusp::complex<double> I(0.0f,1.0f);
cusp::complex<double> zero(0.0f,0.0f);
typedef double    FloatType;
typedef cusp::complex<double>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif


#ifdef FLOAT
typedef float    FloatType;
typedef int      Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<float> I(0.0f,1.0f);
float zero=0;
#endif

#ifdef DOUBLE
typedef double    Scalar;
typedef double    FloatType;
typedef int       Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<double> I(0.0,1.0);
double zero=0;
#endif

template <typename Matrix, typename FloatType>
        void RAMDOS( Matrix& H,typename Matrix::index_type M,typename Matrix::index_type R, const FloatType Emin,const  FloatType Emax,const FloatType alpha,const std::string outputname, int NE ){
            /********************************Algoritmo de Chebyshev y vectores aleatorios ***********************************/
            typedef typename Matrix::value_type Scalar;
            typedef typename Matrix::index_type Integer;
            typedef typename Matrix::value_type::value_type Floating;
            cusp::array1d	<Scalar,cusp::host_memory> mu_h(M,0);
            cusp::array1d	<Scalar,cusp::device_memory> mu;
			chebyshev::random::delta_moments(H,mu_h,M,R);
			mu=mu_h;
			chebyshev::kernel::jackson_kernel(mu,1.0f/(Floating)R); 
			//Floating lambda=3;chebyshev::kernel::lorentz_kernel(mu,1.0f/(Floating)R,lambda);            
			chebyshev::sum::dosram(mu,Emin,Emax,alpha,(Floating)NE,outputname);   
        }

	template <typename Vector, typename Float>  void dosram(Vector& mu,const Float Emin,const  Float Emax,const Float alpha,const Float NE,const std::string outputname){

		typedef typename Vector::value_type Scalar;
		typedef typename Scalar::value_type Floating;
		Float dEn  =2*alpha/NE;
		thrust::counting_iterator<int> index(1);						//Definimos el inidice del vector como iterador
		Floating a=0;
		cusp::complex<Floating> zero=(Floating)0;	
		Scalar DOS;
		dos_binary<Scalar>		delta_chev(a,mu.size(),a);							//Definimos la operacion binaria del kernel (mu,Index)
		thrust::plus<Scalar>	binary_op1;
		//Calculamos el kernel 
		std::ofstream output_file;   output_file.open(outputname.c_str());

		for(Float En=-alpha;En<=alpha;En=En+dEn){
			a=(2/(((Floating)M_PI)*sqrt(1-En*En)))*(2*alpha/(Emax-Emin));
			delta_chev.En=En;
			delta_chev.a =a;
			DOS=thrust::inner_product(mu.begin(),mu.end(), index,zero,binary_op1,delta_chev);	
			output_file<< 0.5*(En*(Emax-Emin)/alpha+(Emax+Emin))<<" "<<DOS.real()<<std::endl;
			} 	
        output_file.close();
        }


int main(int argc, char *argv[])    			//Para simplificar las corridas elegimos N=argv[1], M=argv[2]
{
    
    /***********Declaracion de apuntadores y variables del programa*********/
    int Nx, Ny,D,DD,M, R;	//variables de conteo
	FloatType Emin, Emax,alpha;
	FloatType U;
	int ngpu;
	int SEED;
    int pid=getpid(); // get it as per your OS
	timeval t;
	gettimeofday(&t, NULL);
	std::stringstream ssPID;		//create a stringstream
	ssPID << pid;//add number to the stream
    
    /************Definimos los parametros iniciales **************************/
	if(argc>=8+1){
		Nx=atoi(argv[1]);                                        //Tamaño del sistema
		Ny=atoi(argv[2]);                                        //Tamaño del sistema
		M=atoi(argv[3]);
        U= atof(argv[4]);
        R=atoi(argv[5]);
        //      MachineName=argv[6];
        ngpu=atoi(argv[7]);
        SEED=atoi(argv[8]);
    }else{
        std::cout<<"Numero de parametros erroneos, programa finalizado"<<std::endl;
        std::cout<<"Los parametros son:"<<std::endl;
        std::cout<<"Nx, Ny, M, U ,R  ,MachineName,GPU"<<std::endl;
        return 0;}
	srand(SEED);

    /***********Declaracion de directorios de datos de salida **************/
	std::string dataID("");
	dataID+="Nx";		dataID+=argv[1];
	dataID+="Ny";		dataID+=argv[2];
	dataID+="M";		dataID+=argv[3];
	dataID+="U";		dataID+=argv[4];
	dataID+="R";		dataID+=argv[5];
	dataID+=argv[6] ;

	//Creates the file that will hold the Density of states 
	std::string dospath("data/GrapheneDOS");
	dospath.append(dataID);
	dospath+=".dat";

	//Select the gpu device(Default value 0)
	std::cout<<"Setting device "<<ngpu<<std::endl;
   	hipSetDevice(ngpu);
	D 	=	Nx*Ny;
	DD	=	2*D;
	HCOO H (DD,DD,4*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins

	graphene::lattice(Nx,Ny,H);
	graphene::Anderson(Nx,Ny,H,U);
	RefineSparse(H);
	Emin=-3.0;
	Emax= 3.0;
	alpha=0.9;
	chebyshev::Rescale(H,Emin,Emax,alpha);
	cycletime(-1);
	std::cout<<std::endl<<"Calculating DOS"<<std::endl;
	srand(time(0)*t.tv_usec * t.tv_sec * pid);
	chebyshev::random::DOSRAM(H,M,R,Emin,Emax,alpha,dospath,6556+1);
	cycletime(-1);


    return 0;
}
/***************************REEScalando el hamiltoniano******************/
