#include "utilidades.h"
#include "linear_cuda_utilities.h"
//#include "graphene_cusp.h"
#include "lattice_cusp.h"
#include <cusp/transpose.h>
#include <iostream>
#include <fstream>
#include "kpm_cusp_updated.h"

//#define FCOMPLEX
#define DCOMPLEX
//#define FLOAT
//#define DOUBLE

#ifdef FCOMPLEX
cusp::complex<float> I(0.0f,1.0f);
cusp::complex<float> zero(0.0f,0.0f);
typedef float    FloatType;
typedef cusp::complex<float>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif

#ifdef DCOMPLEX
cusp::complex<double> I(0.0f,1.0f);
cusp::complex<double> zero(0.0f,0.0f);
typedef double    FloatType;
typedef cusp::complex<double>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif


#ifdef FLOAT
typedef float    Scalar	HCOO Vx(DDD,DDD,S*4*DDD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins	
;
typedef float    FloatType;
typedef int      Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<float> I(0.0f,1.0f);
float zero=0;
#endif

#ifdef DOUBLE
typedef double    Scalar;
typedef double    FloatType;
typedef int       Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<double> I(0.0,1.0);
double zero=0;
#endif


int main(int argc, char *argv[])    			//Para simplificar las corridas elegimos N=argv[1], M=argv[2]
{
    
    /***********Declaracion de apuntadores y variables del programa*********/
    int Nx, Ny,D,DD,M, R,ngpu;	//variables de conteo
	FloatType	alpha,W, Emin, Emax,E0min, E0max;
    //srand(1121322113);
    srand(time(0));
    
    /************Definimos los parametros iniciales **************************/
	if(argc>=7+1){
		Nx=atoi(argv[1]);                                        //Tamaño del sistema
		Ny=atoi(argv[2]);                                        //Tamaño del sistema
		M=atoi(argv[3]);
		W=atof(argv[4]);                                     //Densidad de impurezas
        R=atoi(argv[5]);
		//MachineName=argv[6];
        ngpu=atoi(argv[7]);
    }else{
        std::cout<<"Numero de parametros erroneos, programa finalizado"<<std::endl;
        std::cout<<"Los parametros son:"<<std::endl;
        std::cout<<"Nx, Ny, M, W ,R  ,MachineName,GPU"<<std::endl;
        return 0;}

    /***********Declaracion de directorios de datos de salida **************/
	std::string dospath("data/GrapheneDOSKBF");
	dospath+="Nx";		dospath+=argv[1];
	dospath+="Ny";		dospath+=argv[2];
	dospath+="M";		dospath+=argv[3];
	dospath+="W";		dospath+=argv[4];
	dospath+="R";		dospath+=argv[5];
	dospath+=argv[6] ;
	dospath+=".dat"  ;

	std::string sigmaxxpath("data/GrapheneConductivityKBF");
	sigmaxxpath+="Nx"     ; sigmaxxpath+=argv[1];
	sigmaxxpath+="Ny"     ; sigmaxxpath+=argv[2];
	sigmaxxpath+="M"     ; sigmaxxpath+=argv[3];
	sigmaxxpath+="W"     ; sigmaxxpath+=argv[4];
	sigmaxxpath+="R"     ; sigmaxxpath+=argv[5];
	sigmaxxpath+=argv[6] ;
	sigmaxxpath+=".dat"  ;

	std::string dospath0("data/GrapheneDOSLIN");
	dospath0+="Nx";		dospath0+=argv[1];
	dospath0+="Ny";		dospath0+=argv[2];
	dospath0+="M";		dospath0+=argv[3];
	dospath0+="W";		dospath0+=argv[4];
	dospath0+="R";		dospath0+=argv[5];
	dospath0+=argv[6] ;
	dospath0+=".dat"  ;

	std::string sigmaxxpath0("data/GrapheneConductivityLIN");
	sigmaxxpath0+="Nx"     ; sigmaxxpath0+=argv[1];
	sigmaxxpath0+="Ny"     ; sigmaxxpath0+=argv[2];
	sigmaxxpath0+="M"     ; sigmaxxpath0+=argv[3];
	sigmaxxpath0+="W"     ; sigmaxxpath0+=argv[4];
	sigmaxxpath0+="R"     ; sigmaxxpath0+=argv[5];
	sigmaxxpath0+=argv[6] ;
	sigmaxxpath0+=".dat"  ;


    //Select the gpu device(Default value 0)
   hipSetDevice(ngpu);
	D 	=	Nx*Ny;
	DD	=	2*D;
	HCOO H (DD,DD,4*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins
	graphene::lattice(Nx,Ny,H);
	graphene::Anderson(Nx,Ny,H,W);
	HCOO V (DD,DD,4*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins	
	graphene::velocityx(Nx,Ny,H,V);
	RefineSparse(H);
	RefineSparse(V);
	Emin=-3.3;
	Emax= 3.3;
	alpha=0.9;
	chebyshev::Rescale(H,Emin,Emax,alpha);
	E0min=-1;
	E0max= 1;

	chebyshev::random::DOS(H,M,R,Emin,Emax,alpha,dospath,3111);
	cycletime(-1);
	std::cout<<"Calculating SIGMAXx BASTIN"<<std::endl;
	chebyshev::random::SIGMA(H,V,M,R,Emin,Emax,alpha,sigmaxxpath,3511);
	cycletime(-1);
	chebyshev::random::DosConductivity(H,V,M,R,Emin,Emax,alpha,dospath0,sigmaxxpath0,E0min,E0max,100);

    return 0;
}
/***************************REEScalando el hamiltoniano******************/
