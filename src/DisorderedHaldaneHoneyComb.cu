#include "utilidades.h"
#include "linear_cuda_utilities.h"
//#include "graphene_cusp.h"
#include "lattice_cusp.h"
#include <iostream>
#include <fstream>
#include "kpm_cusp.h"

//#define FCOMPLEX
#define DCOMPLEX
//#define FLOAT
//#define DOUBLE

#ifdef FCOMPLEX
cusp::complex<float> I(0.0f,1.0f);
cusp::complex<float> zero(0.0f,0.0f);
typedef float    FloatType;
typedef cusp::complex<float>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif

#ifdef DCOMPLEX
cusp::complex<double> I(0.0f,1.0f);
cusp::complex<double> zero(0.0f,0.0f);
typedef double    FloatType;
typedef cusp::complex<double>    Scalar;
typedef int         Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
#endif


#ifdef FLOAT
typedef float    Scalar	HCOO Vx(DDD,DDD,S*4*DDD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins	
;
typedef float    FloatType;
typedef int      Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<float> I(0.0f,1.0f);
float zero=0;
#endif

#ifdef DOUBLE
typedef double    Scalar;
typedef double    FloatType;
typedef int       Indice;
typedef cusp::coo_matrix<Indice, Scalar, cusp::device_memory> DCOO;
typedef cusp::coo_matrix<Indice, Scalar, cusp::host_memory>   HCOO;
cusp::complex<double> I(0.0,1.0);
double zero=0;
#endif


int main(int argc, char *argv[])    			//Para simplificar las corridas elegimos N=argv[1], M=argv[2]
{
    
    /***********Declaracion de apuntadores y variables del programa*********/
    int Nx, Ny,D,DD,M, R,ngpu;	//variables de conteo
	FloatType   Emin, Emax,E0min, E0max;
	FloatType U,UAB,p,lambda;
    //srand(1121322113);
    srand(time(0));
    
    /************Definimos los parametros iniciales **************************/
	if(argc>=10+1){
		Nx=atoi(argv[1]);                                        //Tamaño del sistema
		Ny=atoi(argv[2]);                                        //Tamaño del sistema
		M=atoi(argv[3]);
        U= atof(argv[4]);
		lambda=atof(argv[5]);
		UAB=atof(argv[6]);
		p=atof(argv[7]);                                     //Densidad de impurezas
        R=atoi(argv[8]);
        //MachineName=argv[9];
        ngpu=atoi(argv[10]);
    }else{
        std::cout<<"Numero de parametros erroneos, programa finalizado"<<std::endl;
        std::cout<<"Los parametros son:"<<std::endl;
        std::cout<<"Nx, Ny, M, U,lambda, UAB ,p ,R  ,MachineName,GPU"<<std::endl;
        return 0;}

    /***********Declaracion de directorios de datos de salida **************/
	std::string dospath("data/GraphenePolarizedSODOS");
	dospath+="Nx";		dospath+=argv[1];
	dospath+="Ny";		dospath+=argv[2];
	dospath+="M";		dospath+=argv[3];
	dospath+="U";		dospath+=argv[4];
	dospath+="lambda";	dospath+=argv[5];
	dospath+="UAB";	dospath+=argv[6];
	dospath+="p";		dospath+=argv[7];
	dospath+="R";		dospath+=argv[8];
	dospath+=argv[9] ;
	dospath+=".dat"  ;

	std::string sigmaxxpath("data/GraphenePolarizedSOConductivityXX");
	sigmaxxpath+="Nx"     ; sigmaxxpath+=argv[1];
	sigmaxxpath+="Ny"     ; sigmaxxpath+=argv[2];
	sigmaxxpath+="M"     ; sigmaxxpath+=argv[3];
	sigmaxxpath+="U"     ; sigmaxxpath+=argv[4];
	sigmaxxpath+="lambda"; sigmaxxpath+=argv[5];
	sigmaxxpath+="UAB"; sigmaxxpath+=argv[6];
	sigmaxxpath+="p"     ; sigmaxxpath+=argv[7];
	sigmaxxpath+="R"     ; sigmaxxpath+=argv[8];
	sigmaxxpath+=argv[9] ;
	sigmaxxpath+=".dat"  ;

	std::string sigmaxypath("data/GraphenePolarizedSOConductivityXY");
	sigmaxypath+="Nx"     ; sigmaxypath+=argv[1];
	sigmaxypath+="Ny"     ; sigmaxypath+=argv[2];
	sigmaxypath+="M"     ; sigmaxypath+=argv[3];
	sigmaxypath+="U"     ; sigmaxypath+=argv[4];
	sigmaxypath+="lambda"	; sigmaxypath+=argv[5];
	sigmaxypath+="UAB"	; sigmaxypath+=argv[6];
	sigmaxypath+="p"     ; sigmaxypath+=argv[7];
	sigmaxypath+="R"     ; sigmaxypath+=argv[8];
	sigmaxypath+=argv[9] ;
	sigmaxypath+=".dat"  ;

	

    //Select the gpu device(Default value 0)
   hipSetDevice(ngpu);
	D 	=	Nx*Ny;
	DD	=	2*D;
    	HCOO H (DD,DD,10*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins
	HCOO Vx(DD,DD,10*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins	
	HCOO Vy(DD,DD,10*DD); // 10 vecinos 3primeros+6segundos+1onsite. Con 2 spins	

	graphene::lattice(Nx,Ny,H);
    graphene::LatticePotential(Nx,Ny,H,(double)1,UAB);
	graphene::Andeson(Nx,Ny,H,U);
	graphene::PolarizedIntrinsecSO(Nx,Ny,H,lambda,p);
	graphene::velocityx(Nx,Ny,H,Vx);
	graphene::velocityy(Nx,Ny,H,Vy);
	RefineSparse(H);
	RefineSparse(Vx);
	RefineSparse(Vy);
//	print(H);
	FindingExtrema(H,Emin,Emax,400);
//	FindingExtrema(H,Emin,Emax);
	E0min=0.97*Emin;
	E0max=0.97*Emax;
//	FindingExtrema(H,Emin,Emax,400);
//	E0min=-3;
//	E0max= 3;	

 	std::cout<<std::endl<<"Calculating DOS"<<std::endl;
	cycletime(-1);
	chebyshev::random::DOS(H,M,R,Emin,Emax,E0min,E0max,dospath,32768); //falta un 8 al final
	cycletime(-1);
	std::cout<<"Calculating SIGMAXY "<<std::endl;
	chebyshev::random::SIGMA(H,Vx,Vy,M,R,Emin,Emax,E0min,E0max,sigmaxypath,16384);
	cycletime(-1);
	std::cout<<"Calculating SIGMAXX "<<std::endl;
	chebyshev::random::SIGMA(H,Vx,Vx,M,R,Emin,Emax,E0min,E0max,sigmaxxpath,16384);//falta un 4 al final


    return 0;
}
/***************************REEScalando el hamiltoniano******************/
	
