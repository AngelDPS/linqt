#include "hip/hip_runtime.h"

#include "full_spectral_sum.hpp"


void 
utility::kernel::JacksonFunction(const my::real m , const my::real M, my::scalar& mu)
{
	const my::real theta= M_PI/(M+1.);
	mu=mu*my::real( 
		  ( 
			(M-m+1.)*
			cos(theta*m) + 
			sin(theta*m)*cos(theta)/sin(theta) 
		  )/(M+1.) );
}; 

void
utility::kernel::LorentzFunction(const my::real,const my::real lambda, 
					const my::real m , 
					const my::real M, my::scalar& mu)
	{
	  mu=mu*my::real( sinh( lambda*(1.-m/M)) /sinh(lambda) );
	}; // end plus




void 
utility::sum::SpectralConductivity(const std::string moment_filename , my::integer M0,  my::integer M1,
						const my::integer NE,
						const std::string output_filename)
{
	if(M1>M0)
	{
		std::cout<<"Warning M1 cannot be larger than M0"<<std::endl;
	}
	std::ifstream moment_file( moment_filename.c_str() );
	std::ofstream output_file( output_filename.c_str() );

	std::string  label_out;
	//Read M from file;
	my::integer M;
	moment_file>>label_out>>M;
	if(M0==0)
		M0=M;
	if(M1==0)
		M1=M;
		
	//Read the bounds and cutoff from file 
	my::real MaxBound, MinBound,  CutOff,norb,spin,area;
	moment_file>>MinBound>>MaxBound>>CutOff>>norb>>spin>>area;
	std::cout<<"The moments will be computed using : M=("<<M0<<","<<M1<<") "<<MinBound<<" "<<MaxBound<<" "<<CutOff<<" "<<norb<<" "<<spin<<" "<<area<<std::endl;
	const my::real Emin=MinBound;
	const my::real Emax=MaxBound;

	//Read the moments from file
	my::real mure, muim;
	my::scalar mutmp;
	my::integer m0,m1;
	thrust::host_vector< thrust_complex > h_mu(M0*M1);

	for(my::integer m=0;m<M;m++)
	for(my::integer n=0;n<M;n++)
	{		
		moment_file>>m0>>m1>>mure>>muim;
		if( m0<M0 && m1< M1)
		{
			mutmp= my::scalar(mure,muim);
			mutmp=mutmp*my::real(4.0);
			if(m0==0)
				mutmp=mutmp*my::real(0.5);
			if(m1==0)
				mutmp=mutmp*my::real(0.5);
			kernel::JacksonFunction(m0,M0,mutmp);
			kernel::JacksonFunction(m1,M1,mutmp);
			h_mu[m0*M0+m1]= thrust_complex( mutmp );
		}
	}
	
	moment_file.close();

	thrust::device_vector< thrust_complex > d_mu( h_mu );
	
	//Creates operatirs  that will be used by the binary_op
	thrust::counting_iterator<my::integer> index(0);						//Definimos el inidice del vector como iterador
	thrust_complex zero= 0.0;	
	thrust::plus<thrust_complex>	binary_sum;
	//The custom binary operator
	SpectralConductivity_binary cond_bin(M0,M1);							//Definimos la operacion binaria del kernel (mu,Index)

//	std::cout<<"Multiply the result for unknown 10 factor"<<std::endl;
	const my::real scal0 = 4*pow(2.*CutOff/(MaxBound-MinBound),1.)/area/spin;	
	const my::real dE =(Emax-Emin)/NE;
	for(my::real E=Emin;E<=Emax;E=E+dE)
	{
		const my::real En   = ( E-(MaxBound+MinBound)*0.5 )*2.*CutOff/(MaxBound-MinBound) ;
		const my::real scal = scal0/pow( 1 - En*En ,2 );

		cond_bin.En=En;
		const my::scalar 
		SIGMA=thrust::inner_product(d_mu.begin(),d_mu.end(), index, zero,binary_sum,cond_bin)*scal;	
		output_file<< E<<" "<<SIGMA.real()<<" "<<SIGMA.imag()<<std::endl;
	} 
	output_file.close();
};
